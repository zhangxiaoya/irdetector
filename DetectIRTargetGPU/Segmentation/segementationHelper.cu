#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
#include "segementationHelper.cuh"
#include "../Assistants/ShowFrame.hpp"
#include "../CCL/MeshCCLKernelD.cuh"
#include "../Models/FourLimits.h"
#include "../Models/Point.h"
#include "../Models/ObjectRect.h"
#include "../Checkers/CheckPerf.h"

void GetAllObjects(int width, int height, int* labelsOnHost, FourLimits* allObjects)
{
	// top
	for(auto r = 0; r < height;++r)
	{
		for(auto c = 0;c < width;++c)
		{
			auto label = labelsOnHost[r * width + c];
			if (allObjects[label].top == -1)
				allObjects[label].top = r;
		}
	}
	// bottom
	for (auto r = height -1; r >= 0; --r)
	{
		for (auto c = 0; c < width; ++c)
		{
			auto label = labelsOnHost[r * width + c];
			if (allObjects[label].bottom == -1)
				allObjects[label].bottom = r;
		}
	}

	// left
	for (auto c = 0; c < width; ++c)
	{
		for (auto r = 0; r < height; ++r)
		{
			auto label = labelsOnHost[r * width + c];
			if (allObjects[label].left == -1)
				allObjects[label].left = c;
		}
	}
	// right
	for (auto c = width -1; c >= 0; --c)
	{
		for (auto r = 0; r < height; ++r)
		{
			auto label = labelsOnHost[r * width + c];
			if (allObjects[label].right == -1)
				allObjects[label].right = c;
		}
	}
}

void do_work(int width, int height, FourLimits* allObjects, ObjectRect* allObjectRects)
{
	for(auto i = 0;i < width * height;++i)
	{
		if (allObjects[i].top == -1)
			continue;
		allObjectRects[i].width = allObjects[i].right - allObjects[i].left + 1;
		allObjectRects[i].height = allObjects[i].bottom - allObjects[i].top + 1;
		allObjectRects[i].lt = Point(allObjects[i].left, allObjects[i].top);
		allObjectRects[i].rb = Point(allObjects[i].right, allObjects[i].bottom);
	}
}

void Segmentation(unsigned char* frameOnHost, int width, int height)
{
	int* labelsOnHost;
	int* labelsOnDevice;
	int* referenceOfLablesOnDevice;
	bool* modificationFlagOnDevice;
	unsigned char* frameOnDevice;

	hipHostMalloc((void**)&labelsOnHost, width * height * sizeof(int));
	hipMalloc((void**)&frameOnDevice, width * height * sizeof(unsigned char));
	hipMalloc((void**)&labelsOnDevice, sizeof(int) * width * height);
	hipMalloc((void**)&referenceOfLablesOnDevice, sizeof(int) * width* height);
	hipMalloc((void**)&modificationFlagOnDevice, sizeof(bool));

	hipMemcpy(frameOnDevice, frameOnHost, sizeof(unsigned char) * width * height, hipMemcpyHostToDevice);

	cv::Mat img;
	ShowFrame::ToMat<unsigned char>(frameOnHost, width, height, img, CV_8UC1);

	ShowFrame::ToTxt<unsigned char>(frameOnHost, "data.txt", width, height);

	CheckPerf(MeshCCL(frameOnDevice, labelsOnDevice, referenceOfLablesOnDevice,modificationFlagOnDevice,width, height),"Mesh CCL");

	hipMemcpy(labelsOnHost, labelsOnDevice, sizeof(int) * width * height, hipMemcpyDeviceToHost);

	ShowFrame::ToTxt<int>(labelsOnHost,"lables.txt", width, height);

	auto allObjects = new FourLimits[width * height];

	CheckPerf(GetAllObjects(width, height, labelsOnHost, allObjects),"All Objects");

	auto allObjectRects = new ObjectRect[width * height];

	CheckPerf(do_work(width, height, allObjects, allObjectRects), "To Rect");

	ShowFrame::DrawRectangles(frameOnHost, allObjectRects, width, height);

	delete[] allObjectRects;
	delete[] allObjects;
	hipHostFree(labelsOnHost);
	hipFree(labelsOnDevice);
	hipFree(referenceOfLablesOnDevice);
	hipFree(frameOnDevice);
	hipFree(modificationFlagOnDevice);
}

#pragma once
#include <hip/hip_runtime.h>

#include <cmath>

typedef unsigned char(*pointFunction_t)(unsigned char, unsigned char);

__device__ unsigned char pComputeMin(unsigned char a, unsigned char b)
{
	return (a < b) ? a : b;
}

__device__ unsigned char pComputeMax(unsigned char a, unsigned char b)
{
	return (a > b) ? a : b;
}

__device__ void FilterStep2K(unsigned char* src, unsigned char* dst, int width, int height, int tile_w, int tile_h, const int radio, const pointFunction_t pPointOperation)
{
	extern __shared__ unsigned char smem[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	auto x = bx * tile_w + tx;
	auto y = by * tile_h + ty - radio;

	smem[ty * blockDim.x + tx] = 0;
	__syncthreads();
	if (x >= width || y < 0 || y >= height)
	{
		return;
	}
	smem[ty * blockDim.x + tx] = src[y * width + x];
	__syncthreads();
	if (y < (by * tile_h) || y >= ((by + 1) * tile_h))
	{
		return;
	}
	auto smem_thread = &smem[(ty - radio) * blockDim.x + tx];
	auto val = smem_thread[0];
#pragma unroll
	for (auto yy = 1; yy <= 2 * radio; yy++)
	{
		val = pPointOperation(val, smem_thread[yy * blockDim.x]);
	}
	dst[y * width + x] = val;
}

__device__ void FilterStep1K(unsigned char* src, unsigned char* dst, int width, int height, int tile_w, int tile_h, const int radio, const pointFunction_t pPointOperation)
{
	extern __shared__ unsigned char smem[];
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	auto x = bx * tile_w + tx - radio;
	auto y = by * tile_h + ty;
	smem[ty * blockDim.x + tx] = 0;
	__syncthreads();
	if (x < 0 || x >= width || y >= height)
	{
		return;
	}
	smem[ty * blockDim.x + tx] = src[y * width + x];
	__syncthreads();
	if (x < (bx * tile_w) || x >= ((bx + 1) * tile_w))
	{
		return;
	}
	auto smem_thread = &smem[ty * blockDim.x + tx - radio];
	auto val = smem_thread[0];
#pragma unroll
	for (auto xx = 1; xx <= 2 * radio; xx++)
	{
		val = pPointOperation(val, smem_thread[xx]);
	}
	dst[y * width + x] = val;
}

__global__ void FilterDStep1(unsigned char* src, unsigned char* dst, int width, int height, int tile_w, int tile_h, const int radio)
{
	FilterStep1K(src, dst, width, height, tile_w, tile_h, radio, pComputeMax);
}

__global__ void FilterDStep2(unsigned char* src, unsigned char* dst, int width, int height, int tile_w, int tile_h, const int radio)
{
	FilterStep2K(src, dst, width, height, tile_w, tile_h, radio, pComputeMax);
}

void FilterDilation(unsigned char* src, unsigned char* dst, int width, int height, int radio)
{
	unsigned char* tempResultOnDevice;
	hipMalloc(&tempResultOnDevice, width * height);

	auto tile_w1 = 256;
	auto tile_h1 = 1;

	dim3 block2(tile_w1 + (2 * radio), tile_h1);
	dim3 grid2(ceil(static_cast<float>(width) / tile_w1), ceil(static_cast<float>(height) / tile_h1));

	auto tile_w2 = 4;
	auto tile_h2 = 64;

	dim3 block3(tile_w2, tile_h2 + (2 * radio));
	dim3 grid3(ceil(static_cast<float>(width) / tile_w2), ceil(static_cast<float>(height) / tile_h2));

	FilterDStep1 <<<grid2, block2, block2.y * block2.x >>>(src, tempResultOnDevice, width, height, tile_w1, tile_h1, radio);
	(hipDeviceSynchronize());
	FilterDStep2 <<<grid3, block3, block3.y * block3.x >>>(tempResultOnDevice, dst, width, height, tile_w2, tile_h2, radio);
	auto cudaerr = hipDeviceSynchronize();
	hipFree(tempResultOnDevice);
}
